// #include <cuda_runtime.h>

#include <hip/hip_runtime.h>
#include <sys/time.h>

#include <stdio.h>
#include <string.h>

#define THREADS 10
#define ROOM_SIZE 10
#define BLOCKS (ROOM_SIZE * ROOM_SIZE + THREADS - 1) / THREADS
#define ITERATION_LIMIT 100

__global__ void simulate_room(float *H) {
	int index = threadIdx.x + blockIdx.x * THREADS;

	int j = index % ROOM_SIZE;
	int i = index / ROOM_SIZE;


	float tmp = H[index];
	for(int it = 0 ; it < ITERATION_LIMIT ; it++) {
		if(i > 0 && i < ROOM_SIZE - 1 && j > 0 && j < ROOM_SIZE - 1)
			tmp = 0.25 * (H[(i - 1) * ROOM_SIZE + j] + H[(i + 1) * ROOM_SIZE + j] + H[i * ROOM_SIZE + j + 1] + H[i * ROOM_SIZE + j - 1]);

		__syncthreads();

		H[index] = tmp;

		__syncthreads();
	}
}

int main(int argc, char* argv[]) {
	float *h_H, *d_H;

	h_H = (float *)malloc(sizeof(float) * ROOM_SIZE * ROOM_SIZE);

	for(int i = 0 ; i < ROOM_SIZE ; i++) {
		for(int j = 0 ; j < ROOM_SIZE ; j++)
			h_H[i * ROOM_SIZE + j] = 0;
	}

	for(int i = 0 ; i < ROOM_SIZE ; i++) {
		h_H[i * ROOM_SIZE + 0] = 20;
		h_H[i * ROOM_SIZE + ROOM_SIZE - 1] = 20;
		h_H[0 * ROOM_SIZE + i] = 20;
		h_H[ROOM_SIZE * (ROOM_SIZE - 1) + i] = 20;
	}

	for(int i = 3 * (ROOM_SIZE) / 10 ; i < 7 * ROOM_SIZE / 10 ; i++)
		h_H[i] = 100;

	hipMalloc((void **) &d_H, sizeof(float) * ROOM_SIZE * ROOM_SIZE);

	// for(int i = 0 ; i < ROOM_SIZE; i++) {
	// 	for(int j = 0 ; j < ROOM_SIZE ; j++)
	// 		printf("%.0f ", h_H[i][j]);
	// 	printf("\n");
	// }

	hipMemcpy(d_H, h_H, sizeof(float) * ROOM_SIZE * ROOM_SIZE, hipMemcpyHostToDevice);
	printf("THREADS %d BLOCKS %d\n", THREADS, BLOCKS);

	struct timeval t1, t2;
	gettimeofday(&t1, 0);

	simulate_room<<<BLOCKS,THREADS>>>(d_H);

	hipDeviceSynchronize();
	gettimeofday(&t2, 0);
	double time1 = (t2.tv_usec-t1.tv_usec);
	printf("Time for GPU:  %.8f us \n", time1);
	hipMemcpy(h_H, d_H, sizeof(float) * ROOM_SIZE * ROOM_SIZE, hipMemcpyDeviceToHost);

	for(int i = 0 ; i < ROOM_SIZE; i++) {
		for(int j = 0 ; j < ROOM_SIZE ; j++)
			printf("%.0f ", h_H[i * ROOM_SIZE + j]);
		printf("\n");
	}


	// for(int i = 0 ; i < ROOM_SIZE ; i++)

}