#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <time.h>
#include <stdio.h>
#include <string.h>

#define THREADS 64
#define MAX_VALUE 10000
#define ll long long
#define D_SIZE 1000

__global__ void sum_array(ll *a_1, ll *a_2, ll *a_3, ll *a_4, ll *b_sum, ll *b_sq,ll *b_max, ll *b_min, ll N) {
	int index = threadIdx.x + blockIdx.x * THREADS;
	int local_index = threadIdx.x;

	// MAX ELEMENT COMPUTATION
	for(unsigned int width = THREADS / 2; width > 0; width /= 2) {
		if(local_index < width && index + width < N) {
			a_1[index] = max(a_1[index], a_1[index + width]);
		}
		__syncthreads();
	}	
	if(threadIdx.x == 0) b_max[blockIdx.x] = a_1[index];

	// MIN ELEMENT COMPUTATION
	for(unsigned int width = THREADS / 2; width > 0; width /= 2) {
		if(local_index < width && index + width < N) {
			a_2[index] = min(a_2[index], a_2[index + width]);
		}
		__syncthreads();
	}
	if(threadIdx.x == 0) b_min[blockIdx.x] = a_2[index];

	// SUM ELEMENT COMPUTATION
	for(unsigned int width = THREADS / 2; width > 0; width /= 2) {
		if(local_index < width && index + width < N) {
			a_3[index] += a_3[index + width];
		}
		__syncthreads();
	}
	if(threadIdx.x == 0) b_sum[blockIdx.x] = a_3[index];

	// STANDARD DEVIATION
	for(unsigned int width = THREADS / 2; width > 0; width /= 2) {
		if(local_index < width && index + width < N) {
			a_4[index] += a_4[index + width];
		}
		__syncthreads();
	}	
	if(threadIdx.x == 0) b_sq[blockIdx.x] = a_4[index];
}

int main(int argc, char* argv[]) {
	ll *a, *a_sq, *h_sq, *h_sum, *h_max, *h_min; 		
	ll *d_a, *d_a2, *d_a3, *d_a4, *d_sq, *d_sum, *d_max, *d_min; 
	ll N;
	struct timeval t1, t2;


	if(argc <= 1) {
		printf("No argument provided. Taking N = 1e8 \n");
		N = 100000000;
	}
	else {
		int val = (int)atoi(argv[1]);
		if(val == 0) {
			printf("Couldn't interpret N. Taking N = 1e8 \n");
			N = 100000000;
		}
		else
			N = (ll)val;
	}
	ll BLOCKS = (N + THREADS - 1)/THREADS;
	ll array_size = N * sizeof(ll);

	ll blocksize = sizeof(ll) * BLOCKS;

	hipMalloc((void **) &d_a, array_size);
	hipMalloc((void **) &d_a2, array_size);
	hipMalloc((void **) &d_a3, array_size);
	hipMalloc((void **) &d_a4, array_size);
	hipMalloc((void **) &d_sum, blocksize);
	hipMalloc((void **) &d_max, blocksize);
	hipMalloc((void **) &d_min, blocksize);
	hipMalloc((void **) &d_sq, blocksize);

	a = (ll *)malloc(array_size);
	a_sq = (ll *)malloc(array_size);
	h_sq = (ll *)malloc(blocksize);
	h_max = (ll *)malloc(blocksize);
	h_min = (ll *)malloc(blocksize);
	h_sum = (ll *)malloc(blocksize);

	for(int i = 0 ; i < N ; i++) {
		a[i] = rand() % MAX_VALUE;
		a_sq[i] = a[i] * a[i];
	}

	// Copy inputs to GPU
	hipMemcpy(d_a, a, N * sizeof(ll), hipMemcpyHostToDevice);
	hipMemcpy(d_a2, a, N * sizeof(ll), hipMemcpyHostToDevice);
	hipMemcpy(d_a3, a, N * sizeof(ll), hipMemcpyHostToDevice);
	hipMemcpy(d_a4, a_sq, N * sizeof(ll), hipMemcpyHostToDevice);

	gettimeofday(&t1, 0);

	sum_array<<<BLOCKS,THREADS>>>(d_a, d_a2, d_a3, d_a4, d_sum, d_sq, d_max, d_min, N);

	hipDeviceSynchronize();
	gettimeofday(&t2, 0);
	double time1 = (t2.tv_usec-t1.tv_usec);
	printf("Time for GPU:  %.8f us \n", time1);


	// Copy result back to host
	hipMemcpy(h_sq, d_sq, blocksize, hipMemcpyDeviceToHost);
	hipMemcpy(h_max, d_max, blocksize, hipMemcpyDeviceToHost);
	hipMemcpy(h_min, d_min, blocksize, hipMemcpyDeviceToHost);
	hipMemcpy(h_sum, d_sum, blocksize, hipMemcpyDeviceToHost);

	ll max_arr, min_arr;
	double mean, std_dev;
	mean = 0;
	max_arr = -1e9;
	min_arr = 1e9;
	std_dev = 0;
	for(int i = 0 ; i < BLOCKS ; i++) {
		mean += h_sum[i];
		max_arr = max(max_arr, h_max[i]);
		min_arr = min(min_arr, h_min[i]);
		std_dev += h_sq[i];
	}
	mean /= N;
	std_dev = (std_dev / N) - mean * mean;
	printf("Parallel : \n");
	printf("Max : %lld, Min : %lld, Mean : %lf, Standard Deviation : %lf\n", max_arr, min_arr, mean, std_dev);
	printf("\n");

	ll max_s, min_s;
	double mean_s, std_dev_s;
	max_s = -1e9;
	min_s = 1e9;
	mean_s = std_dev_s = 0;


	// Serial Implementation

	gettimeofday(&t1, 0);
	for(int i = 0 ; i < N ; i++) {
		mean_s += a[i];
		std_dev_s += a[i] * a[i];
		max_s = max(max_s, a[i]);
		min_s = min(min_s, a[i]);
	}
	mean_s /= N;
	std_dev_s = (std_dev_s / N) - mean * mean;

	gettimeofday(&t2, 0);
	double time2 = (t2.tv_usec-t1.tv_usec);
	printf("Time for CPU:  %.8f us \n", time2);

	printf("Serial : \n");
	printf("Max : %lld, Min : %lld, Mean : %lf, Standard Deviation : %lf\n", max_s, min_s, mean_s, std_dev_s);
	printf("\n");

	printf("Speedup = %f\n", time2/time1);

	// End Serial Implementation

	// Cleanup
	free(a);
	free(a_sq);
	free(h_sq); 
	free(h_max); 
	free(h_min); 
	free(h_sum); 

	// Cuda free
	hipFree(d_a);
	hipFree(d_a2);
	hipFree(d_a3);
	hipFree(d_a4);
	hipFree(d_sq);
	hipFree(d_max);
	hipFree(d_min);
	hipFree(d_sum);

	return 0;
}